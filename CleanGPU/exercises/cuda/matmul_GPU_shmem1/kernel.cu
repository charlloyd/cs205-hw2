#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "hipblas.h"
#include "../debug.h"

typedef float floatType_t;

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

#define SIZE 1024

/* setup various hard-coded parameters for this kernel */

#define TBX 64 // Size of C this CTA is responsible for, x dimension
#define TBY 64 // Size of C this CTA is responsible for, y dimension
#define TX 16 // Thread block size, x dimension
#define TY 16 // Thread block size, y dimension
#define BK 16 // square block of K size
#define NX 4  // = TBX/TX == number of iterations to do TBX work with TX blocks
#define NY 4  // = TBY/TY == number of iterations to do TBY work with TY blocks

__global__ void GPU_shmem2(const int m, floatType_t const * const a, 
     floatType_t const * const b, floatType_t *c )
{

/* setup some constants for later use */

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int iby = blockIdx.y * TBY;
	const int ibx = blockIdx.x * TBX;

/* shared memory arrays for A and B */

        /* insert code for shared mem array sizes */
	__shared__ floatType_t as[ FIXME ][ FIXME ];
	__shared__ floatType_t bs[ FIXME ][ FIXME ];
	
/* space for C to be held in registers */

        /* insert code for c_tmp size */ 
	floatType_t c_tmp[ FIXME ][ FIXME ] ;

	/* zero the temp C array */

#pragma unroll
        /* complete the upper limit of the for loops */
	for ( int i = 0 ; i < FIXME ; i++) { 
		for ( int j = 0 ; j < FIXME ; j++) {
			c_tmp[i][j] = 0.0;
		}
	}

	/* calculate my initial offset into A and B */

	int aoff = INDX( ibx + tx, ty, m );
	int boff = INDX( tx, iby + ty, m );

	/* main loop over blocks of K */

	for( int Kblock = 0; Kblock < m; Kblock+=BK )
	{

		/* read block of A into shared memory */

#pragma unroll
		for ( int i = 0; i < NX ; i ++ ) 
		{
                        /* complete the index into the array */
			as[ FIXME ][ FIXME ] = a[ (aoff + FIXME) ];
		}

		/* read block of B into shared memory */

#pragma unroll
		for ( int i = 0; i < NY ; i ++ ) 
		{
                        /* complete the index into the arrays */
			bs[ FIXME ][ FIXME ] = b[ (boff + FIXME) ];
		}


		/* increment A and B offsets  for next round of data reads */

		boff += BK;
		aoff += m * BK;

		/* triply nested loop to perform the matmult on the blocks */

#pragma unroll
                /* insert code to complete the loop bounds for j and i */
		for( int k = 0 ; k < BK ; k++ )
		{
#pragma unroll
			for (int j = 0 ; j < FIXME ; j++ )
			{
#pragma unroll
				for (int i = 0 ; i < FIXME ; i++ )
				{
                                        /* insert code to complete the matrix multiply */
					c_tmp[ i ][ j ] += as[ tx + TX*i ][ k ] * bs[ k ][ ty + j*TY ];
				}
			}
		}

	} /* end for Kblock */

	/* set coff to its proper index int the C matrix */

        /* insert code to set coff to its proper location in the C matrix */
	int coff = INDX( FIXME, FIXME, m );
  
	/* write results to the C matrix */

#pragma unroll
	for ( int j = 0 ; j < FIXME ; j++ ) 
	{
#pragma unroll
		for ( int i = 0 ; i < FIXMe ; i++ )
		{      
                        /* insert code to write c_tmp elements to global C matrix */
			c[ coff + INDX( FIXME, FIXME, m )] = c_tmp[FIXME][FIXME];
		}
	}
 
} /* end GPU_shmem1 */

int main( int argc, char *argv[] )
{

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    floatType_t *h_a, *h_b, *h_c, *h_c1;
    floatType_t *d_a, *d_b, *d_c;
 
    size_t numbytes = (size_t ) size * (size_t ) size * sizeof( floatType_t );

    h_a = (floatType_t *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (floatType_t *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (floatType_t *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	h_c1 = (floatType_t *) malloc( numbytes );
    if( h_c1 == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	/* zero out the host memory for C matrices */

    memset( h_c, 0, numbytes );
    memset( h_c1, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

	/* initialize the A and B matrices */

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

	/* allocate a, b, c in gpu memory */

    checkCUDA( hipMalloc( (void **)&d_a, numbytes ) );
    checkCUDA( hipMalloc( (void **)&d_b, numbytes ) );
    checkCUDA( hipMalloc( (void **)&d_c, numbytes ) );
	
	/* copy a and b to device */

    checkCUDA( hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice ) );
    checkCUDA( hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice ) );

    hipblasHandle_t handle;
    checkCUBLAS( hipblasCreate( &handle ) );

    floatType_t alpha = 1.0;
    floatType_t beta  = 0.0;

	/* start timers */

    hipEvent_t start, stop;
    checkCUDA( hipEventCreate( &start ) );
    checkCUDA( hipEventCreate( &stop ) );
    checkCUDA( hipEventRecord( start, 0 ) );

	/* call CUBLAS dgemm */

    if( sizeof( floatType_t ) == 4 )
    {
checkCUBLAS( 
hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 (float *)&alpha, 
                 (float *)d_a, size,
                 (float *)d_b, size,
                 (float *)&beta,
                 (float *)d_c, size )
            );
    } /* end if */
    else
    {
checkCUBLAS( 
hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 (double *)&alpha, 
                 (double *)d_a, size,
                 (double *)d_b, size,
                 (double *)&beta,
                 (double *)d_c, size )
            );
    } /* end else */

	/* stop timers */

    checkCUDA( hipEventRecord( stop, 0 ) );
    checkCUDA( hipEventSynchronize( stop ) );
    float elapsedTime;
    checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

	/* print GPU CUBLAS timing information */

    fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
	/* copy C from device to host for error checking */

    checkCUDA( hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost ) );

	/* reset C on device to zero */

	checkCUDA( hipMemset( d_c, 0, numbytes ) );

	/* setup grid and block sizes */

	dim3 threads( TX, TY, 1 );
	dim3 blocks( size / ( TBX ), size / ( TBY ), 1 );	

	/* call GPU_naive */

	printf("block.X %d block.Y %d\n",blocks.x, blocks.y );
	printf("threads.x %d threads.y %d\n",threads.x, threads.y );
    
/* start timers */

	checkCUDA( hipEventRecord( start, 0 ) );

/* call the kernel */

	GPU_shmem2<<< blocks, threads >>> ( size, d_a, d_b, d_c );
        checkKERNEL()

	/* stop timers */

    checkCUDA( hipEventRecord( stop, 0 ) );
    checkCUDA( hipEventSynchronize( stop ) );
	elapsedTime = 0.0f;
    checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

	/* print data for GPU naive */

    fprintf(stdout, "Total time GPU SHMEM is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
	/* copy C back to host */
	
	checkCUDA( hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost ) );

    checkCUBLAS( hipblasDestroy( handle ) );
    checkCUDA( hipEventDestroy( start ) );
    checkCUDA( hipEventDestroy( stop ) );

/* check CUBLAS versus GPU NAIVE numerical results */

    double temp = 0.0;

    for( int i = 0; i < size * size; i++ )
    {
       temp = max( temp, abs( (double)h_c[i] - (double)h_c1[i] )/
                      abs((double)h_c[i]) );
    } /* end for */
    printf("Maximum error is %e percent \n",temp*100.0);
    if( temp > 0.001 ) printf("FAIL\n");
    else printf("PASS\n");

/* cleanup */


    checkCUDA( hipFree( d_a ) );
    checkCUDA( hipFree( d_b ) );
    checkCUDA( hipFree( d_c ) );

    free( h_a );
    free( h_b );
    free( h_c );
    free( h_c1 );

    checkCUDA( hipDeviceReset() );

    return 0;
}
