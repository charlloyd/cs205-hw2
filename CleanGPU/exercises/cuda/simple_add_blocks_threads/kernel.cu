#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "../debug.h"

__global__ void add(int *a, int *b, int *c)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof( int );

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

/* allocate space for device copies of a, b, c */

  checkCUDA( hipMalloc( (void **) &d_a, size ) );
  checkCUDA( hipMalloc( (void **) &d_b, size ) );
  checkCUDA( hipMalloc( (void **) &d_c, size ) );

/* allocate space for host copies of a, b, c and setup input values */

  a = (int *)malloc( size );
  b = (int *)malloc( size );
  c = (int *)malloc( size );

  for( int i = 0; i < N; i++ )
  {
    a[i] = b[i] = i;
    c[i] = 0;
  }

/* copy inputs to device */

  checkCUDA( hipMemcpy( d_a, a, size, hipMemcpyHostToDevice ) );
  checkCUDA( hipMemcpy( d_b, b, size, hipMemcpyHostToDevice ) );

/* zero out the C array */

  checkCUDA( hipMemset( d_c, 0, size ) );

/* launch the kernel on the GPU */
/* insert the launch parameters to launch properly using blocks and threads */
  add<<< FIXME, FIXME >>>( d_a, d_b, d_c );
  checkKERNEL()

/* copy result back to host */

  checkCUDA( hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost ) );

  int success = 1;

  for( int i = 0; i < N; i++ )
  {
    if( c[i] != a[i] + b[i] )
    {
      printf("c[%d] = %d\n",i,c[i] );
      success = 0;
      break;
    } /* end if */
  }

  if( success == 1 ) printf("PASS\n");
  else               printf("FAIL\n");

/* clean up */

  free(a);
  free(b);
  free(c);
  checkCUDA( hipFree( d_a ) );
  checkCUDA( hipFree( d_b ) );
  checkCUDA( hipFree( d_c ) );

  checkCUDA( hipDeviceReset() );
	
  return 0;
} /* end main */
