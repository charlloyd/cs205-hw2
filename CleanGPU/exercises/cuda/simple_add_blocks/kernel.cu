#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "../debug.h"

__global__ void add(int *a, int *b, int *c)
{
/* add the proper index so each block calculates a different value in the 
   array  */
  c[FIXME] = a[FIXME] + b[FIXME];
}

#define N 32

int main()
{
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof( int );

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

/* allocate space for device copies of a, b, c */

  checkCUDA( hipMalloc( (void **) &d_a, size ) );
/* insert code here for d_b and d_c */
  FIXME

/* allocate space for host copies of a, b, c and setup input values */

  a = (int *)malloc( size );
  b = (int *)malloc( size );
  c = (int *)malloc( size );

  for( int i = 0; i < N; i++ )
  {
    a[i] = b[i] = i;
    c[i] = 0;
  } /* end for */

/* copy inputs to device */

  checkCUDA( hipMemcpy( d_a, a, size, hipMemcpyHostToDevice ) );
/* insert code to copy b to the device */
  FIXME

/* zero out C array */

  checkCUDA( hipMemset( d_c, 0, size ) );

/* launch the kernel on the GPU */
/* finish the kernel launch with N blocks and 1 thread per block */
  add<<< FIXME, FIXME >>>( d_a, d_b, d_c );
  checkKERNEL()

/* copy result back to host */

  checkCUDA( hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost ) );

  int success = 1;

  for( int i = 0; i < N; i++ )
  {
    printf("c[%d] = %d\n",i,c[i]);
    if( c[i] != a[i] + b[i] )
    {
      success = 0;
      break;
    } /* end if */
  } /* end for */

  if( success == 1 ) printf("PASS\n");
  else               printf("FAIL\n");

/* clean up */

  free(a);
  free(b);
  free(c);
  checkCUDA( hipFree( d_a ) );
  checkCUDA( hipFree( d_b ) );
  checkCUDA( hipFree( d_c ) );
	
  return 0;
} /* end main */
