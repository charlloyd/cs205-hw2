#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "../debug.h"

#define N ( 1 << 27 )
#define THREADS_PER_BLOCK 256

/* only works for float currently.  double will break this code due to 
   lack of 64bit floating point atomics
 */

#define FLOATTYPE_T float

/* sumReduction kernel using atomics */

__global__ void sumReduction(int n, FLOATTYPE_T *in, FLOATTYPE_T *sum)
{
/* calculate global index in the array */
  int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
/* return if my global index is larger than the array size */
  if( globalIndex >= n ) return;

/* grid stride loop where array is larger than number of threads
 * launched, using atomics
 */

  FLOATTYPE_T temp = 0.0;

  for( int i = globalIndex; i < n; i += blockDim.x * gridDim.x )
  {
      temp += in[i];
  } /* end for */

  atomicAdd( sum, temp );

  return;
}

int main()
{
  FLOATTYPE_T *h_in, h_sum, cpu_sum;
  FLOATTYPE_T *d_in, *d_sum;
  int size = N;
  int memBytes = size * sizeof( FLOATTYPE_T );

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

/* allocate space for device copies of in, out */

  checkCUDA( hipMalloc( &d_in, memBytes ) );
  checkCUDA( hipMalloc( &d_sum, sizeof(FLOATTYPE_T) ) );

/* allocate space for host copies of in, out and setup input values */

  h_in = (FLOATTYPE_T *)malloc( memBytes );

  for( int i = 0; i < size; i++ )
  {
    h_in[i] = FLOATTYPE_T( rand() ) / ( FLOATTYPE_T (RAND_MAX) + 1.0 );
    if( i % 2 == 0 ) h_in[i] = -h_in[i];
  }

  h_sum      = 0.0;
  cpu_sum   = 0.0;

/* copy inputs to device */

  checkCUDA( hipMemcpy( d_in, h_in, memBytes, hipMemcpyHostToDevice ) );
  checkCUDA( hipMemset( d_sum, 0, sizeof(FLOATTYPE_T) ) );

/* calculate block and grid sizes */

  dim3 threads( THREADS_PER_BLOCK, 1, 1);
  
/* choose blocksize such that it will be smaller than the max that this
   GPU allows */ 
 
  int blk = 64;
  dim3 blocks( blk, 1, 1);

/* start the timers */

  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );
  checkCUDA( hipEventRecord( start, 0 ) );

/* launch the kernel on the GPU */

  sumReduction<<< blocks, threads >>>( size, d_in, d_sum );
  checkKERNEL()

/* stop the timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  float elapsedTime;
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print GPU perf data */

  printf("Total elements is %d, %f GB\n", size, sizeof(FLOATTYPE_T)*
    (double)size * 1.e-9 );
  printf("GPU total time is %f ms, bandwidth %f GB/s\n", elapsedTime,
    sizeof(FLOATTYPE_T) * (double) size /
    ( (double) elapsedTime / 1000.0 ) * 1.e-9);

/* copy result back to host */

  checkCUDA( hipMemcpy( &h_sum, d_sum, sizeof(FLOATTYPE_T), 
    hipMemcpyDeviceToHost ) );

/* calculate CPU results */

  checkCUDA( hipEventRecord( start, 0 ) );

  for( int i = 0; i < size; i++ )
  {
    cpu_sum += h_in[i];
  } /* end for */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print CPU perf data */

  printf("CPU total time is %f ms, bandwidth %f GB/s\n", elapsedTime,
    sizeof(FLOATTYPE_T) * (double) size /
    ( (double) elapsedTime / 1000.0 ) * 1.e-9);

/* calculate error */

  FLOATTYPE_T diff = abs( cpu_sum - h_sum );

  if( diff / abs(h_sum) < 0.001 ) printf("PASS\n");
  else
  {                       
    printf("FAIL\n");
    printf("Error is %f\n", diff / h_sum );
  } /* end else */

/* clean up */

  free(h_in);
  checkCUDA( hipFree( d_in ) );
  checkCUDA( hipFree( d_sum ) );

  checkCUDA( hipDeviceReset() );
	
  return 0;
} /* end main */
