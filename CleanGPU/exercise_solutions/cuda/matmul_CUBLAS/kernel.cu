/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "hipblas.h"
#include "../debug.h"

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

#define SIZE 1024

void host_dgemm( int m, int n, int k, double *a, double *b, double *c )
{

  for( int j = 0; j < n; j++ )
  {
    for( int i = 0; i < m; i++ )
    {
      for( int koff = 0; koff < k; koff++ )
      {
	c[INDX(i, j, m)] += a[INDX( i, koff, m )] * b[INDX( koff, j, n )];
      } /* end for i */
    } /* end jb */
  } /* end for j */

} /* end host_dgemm */

int main( int argc, char *argv[] )
{

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    double *h_a, *h_b, *h_c, *h_cdef;
    double *d_a, *d_b, *d_c;
 
    size_t numbytes = size * size * sizeof( double );

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (double *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (double *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_cdef = (double *) malloc( numbytes );
    if( h_cdef == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    memset( h_c, 0, numbytes );
    memset( h_cdef, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

    checkCUDA( hipMalloc( (void **)&d_a, numbytes ) );
    checkCUDA( hipMalloc( (void **)&d_b, numbytes ) );
    checkCUDA( hipMalloc( (void **)&d_c, numbytes ) );

    hipEvent_t start, stop;
    checkCUDA( hipEventCreate( &start ) );
    checkCUDA( hipEventCreate( &stop ) );


    checkCUDA( hipEventRecord( start, 0 ) );

    host_dgemm( size, size, size, h_a, h_b, h_cdef );

    checkCUDA( hipEventRecord( stop, 0 ) );
    checkCUDA( hipEventSynchronize( stop ) );
    float elapsedTime;
    checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

    fprintf(stdout, "Total time CPU is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );


    checkCUDA( hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice ) );
    checkCUDA( hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice ) );

    hipblasHandle_t handle;
    checkCUBLAS( hipblasCreate( &handle ) );

    double alpha = 1.0;
    double beta  = 0.0;

    checkCUDA( hipEventRecord( start, 0 ) );

    checkCUBLAS( hipblasSetStream( handle, 0 ) );

    checkCUBLAS( 
    hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha, 
                 d_a, size,
                 d_b, size,
                 &beta,
                 d_c, size )
               );


    checkCUDA( hipEventRecord( stop, 0 ) );
    checkCUDA( hipEventSynchronize( stop ) );
    checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

    fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", 
            elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
    checkCUDA( hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost ) );

    checkCUBLAS( hipblasDestroy( handle ) );
    checkCUDA( hipEventDestroy( start ) );
    checkCUDA( hipEventDestroy( stop ) );

    double temp = 0.0;

    for( int i = 0; i < size * size; i++ )
    {
      temp += ( h_c[i] - h_cdef[i] ) * ( h_c[i] - h_cdef[i] );
    } /* end for */
    printf("error is %f\n",temp);
    if( temp > 10 ) printf("FAIL\n");
    else printf("PASS\n");

    checkCUDA( hipFree( d_a ) );
    checkCUDA( hipFree( d_b ) );
    checkCUDA( hipFree( d_c ) );

    free( h_a );
    free( h_b );
    free( h_c );
    free( h_cdef );

    checkCUDA( hipDeviceReset() );
    return 0;
}
