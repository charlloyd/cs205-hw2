#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "hipblas.h"
#include "../debug.h"

typedef float floatType_t;

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

/* naive GPU kernel where each element of C is computed by a single thread */

__global__ void GPU_naive( const int m, floatType_t const * const a, 
     floatType_t const * const b, floatType_t * const c )
{

/* determine my threads's row and col indices in the global C matrix */

  const int myrow = blockDim.x * blockIdx.x + threadIdx.x;
  const int mycol = blockDim.y * blockIdx.y + threadIdx.y;

/* if my row and col are in the C matrix, then calculate that value of C */

  if( myrow < m && mycol < m )
  {
    register floatType_t temp = 0.0;

    for( int k = 0; k < m; k++ ) 
      temp += a[INDX( myrow, k, m )] * b[INDX( k, mycol, m )];

    c[INDX( myrow, mycol, m )] = temp;
  } /* end if */

	return;
} /* end GPU_naive */

int main( int argc, char *argv[] )
{

/* get GPU device number and name */
  const int SIZE[3] =  {pow(2,6), pow(2,10), pow(2,16)};
for(size_t s = 0; s < 3; s++){
  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );


  const int size = SIZE[s];

  fprintf(stdout, "Matrix size is %d\n",size);

  floatType_t *h_a, *h_b, *h_c, *h_c1;
  floatType_t *d_a, *d_b, *d_c;
 
  size_t numbytes = (size_t ) size * (size_t ) size * sizeof( floatType_t );

  h_a = (floatType_t *) malloc( numbytes );
  if( h_a == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_b = (floatType_t *) malloc( numbytes );
  if( h_b == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_c = (floatType_t *) malloc( numbytes );
  if( h_c == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_c1 = (floatType_t *) malloc( numbytes );
  if( h_c1 == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

/* zero out the host memory for C matrices */

  memset( h_c, 0, numbytes );
  memset( h_c1, 0, numbytes );

  fprintf( stdout, "Total memory required is %lf MB\n", 
     3.0 * (double) numbytes / 1000000.0 );

/* initialize the A and B matrices */

  for( int i = 0; i < size * size; i++ )
  {
    h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
  }

/* allocate a, b, c in gpu memory */

  checkCUDA( hipMalloc( (void **)&d_a, numbytes ) );
  checkCUDA( hipMalloc( (void **)&d_b, numbytes ) );
  checkCUDA( hipMalloc( (void **)&d_c, numbytes ) );
	
/* copy a and b to device */

  checkCUDA( hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice ) );
  checkCUDA( hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice ) );

  hipblasHandle_t handle;
  checkCUBLAS( hipblasCreate( &handle ) );

  floatType_t alpha = 1.0;
  floatType_t beta  = 0.0;

/* start timers */

  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );
  checkCUDA( hipEventRecord( start, 0 ) );

/* call CUBLAS dgemm */

  if( sizeof( floatType_t ) == 4 ) 
  {
  checkCUBLAS( 
  hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               size, size, size,
               (float *)&alpha, 
               (float *)d_a, size,
               (float *)d_b, size,
               (float *)&beta,
               (float *)d_c, size )
              );
  } /* end if */
  else
  {
  checkCUBLAS( 
  hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               size, size, size,
               (double *)&alpha, 
               (double *)d_a, size,
               (double *)d_b, size,
               (double *)&beta,
               (double *)d_c, size )
              );
  } /* end else */

/* stop timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  float elapsedTime;
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print GPU CUBLAS timing information */

  fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GFlop/s\n", 
    2.0 * (double) size * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
/* copy C from device to host for error checking */

  checkCUDA( hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost ) );

/* reset C on device to zero */

  checkCUDA( hipMemset( d_c, 0, numbytes ) );

/* setup grid and block sizes */

  dim3 threads( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1 );
  dim3 blocks( size / THREADS_PER_BLOCK_X + 1, 
               size / THREADS_PER_BLOCK_Y + 1, 1 );

/* start timers */

  checkCUDA( hipEventRecord( start, 0 ) );

/* call GPU_naive */

  GPU_naive<<< blocks, threads >>> ( size, d_a, d_b, d_c );
  checkKERNEL()

/* stop timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print data for GPU naive */

  fprintf(stdout, "Total time GPU NAIVE is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GFlop/s\n", 
    2.0 * (double) size * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
/* copy C back to host */
	
  checkCUDA( hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost ) );

  checkCUBLAS( hipblasDestroy( handle ) );
  checkCUDA( hipEventDestroy( start ) );
  checkCUDA( hipEventDestroy( stop ) );

/* check CUBLAS versus GPU NAIVE numerical results */

  double temp = 0.0;

  for( int i = 0; i < size * size; i++ )
  {
     temp = max( temp, abs( (double)h_c[i] - (double)h_c1[i] )/
                      abs((double)h_c[i]) );
  } /* end for */
  printf("Maximum error is %e percent \n",temp*100.0);
  if( temp > 0.001 ) printf("FAIL\n");
  else printf("PASS\n");

/* cleanup */

  checkCUDA( hipFree( d_a ) );
  checkCUDA( hipFree( d_b ) );
  checkCUDA( hipFree( d_c ) );

  free( h_a );
  free( h_b );
  free( h_c );
  free( h_c1 );

  checkCUDA( hipDeviceReset() );
}

  return 0;
}
