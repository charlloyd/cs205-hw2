#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include <math.h>
#include "../debug.h"

/* definitions of threadblock size in X and Y directions */

#define THREADS_PER_BLOCK_X 32
#define THREADS_PER_BLOCK_Y 32

/* definition of matrix linear dimension */

#define SIZE 4096

/* macro to index a 1D memory array with 2D indices in column-major order */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* CUDA kernel for shared memory matrix transpose */

__global__ void smem_cuda_transpose( const int m, 
                                     double const * const a, 
                                     double * const c )
{
	
/* declare a shared memory array */

  __shared__ double smemArray[THREADS_PER_BLOCK_X][THREADS_PER_BLOCK_Y+1];
	
/* determine my row and column indices for the error checking code */

  const int myRow = blockDim.x * blockIdx.x + threadIdx.x;
  const int myCol = blockDim.y * blockIdx.y + threadIdx.y;

/* determine my row tile and column tile index */

  const int tileX = blockDim.x * blockIdx.x;
  const int tileY = blockDim.y * blockIdx.y;

  if( myRow < m && myCol < m )
  {
/* read to the shared mem array */
/* HINT: threadIdx.x should appear somewhere in the first argument to */
/* your INDX calculation for both a[] and c[].  This will ensure proper */
/* coalescing. */

   smemArray[threadIdx.x][threadIdx.y] = 
      a[INDX( tileX + threadIdx.x, tileY + threadIdx.y, m )];
  } /* end if */

/* synchronize */
  __syncthreads();
		
  if( myRow < m && myCol < m )
  {
/* write the result */
    c[INDX( tileY + threadIdx.x, tileX + threadIdx.y, m )] = 
           smemArray[threadIdx.y][threadIdx.x];
  } /* end if */
  return;

} /* end smem_cuda_transpose */

void host_transpose( const int m, double const * const a, double * const c )
{
	
/* 
 *  naive matrix transpose goes here.
 */
 
  for( int j = 0; j < m; j++ )
  {
    for( int i = 0; i < m; i++ )
    {
      c[INDX(i,j,m)] = a[INDX(j,i,m)];
    } /* end for i */
  } /* end for j */

} /* end host_dgemm */

int main( int argc, char *argv[] )
{

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

  int size = SIZE;

  fprintf(stdout, "Matrix size is %d\n",size);

/* declaring pointers for array */

  double *h_a, *h_c;
  double *d_a, *d_c;
 
  size_t numbytes = (size_t) size * (size_t) size * sizeof( double );

/* allocating host memory */

  h_a = (double *) malloc( numbytes );
  if( h_a == NULL )
  {
    fprintf(stderr,"Error in host malloc h_a\n");
    return 911;
  }

  h_c = (double *) malloc( numbytes );
  if( h_c == NULL )
  {
    fprintf(stderr,"Error in host malloc h_c\n");
    return 911;
  }

/* allocating device memory */

  checkCUDA( hipMalloc( (void**) &d_a, numbytes ) );
  checkCUDA( hipMalloc( (void**) &d_c, numbytes ) );

/* set result matrices to zero */

  memset( h_c, 0, numbytes );
  checkCUDA( hipMemset( d_c, 0, numbytes ) );

  fprintf( stdout, "Total memory required per matrix is %lf MB\n", 
     (double) numbytes / 1000000.0 );

/* initialize input matrix with random value */

  for( int i = 0; i < size * size; i++ )
  {
    h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
  } /* end for */

/* copy input matrix from host to device */

  checkCUDA( hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice ) );

/* create and start timer */

  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );
  checkCUDA( hipEventRecord( start, 0 ) );

/* call naive cpu transpose function */

  host_transpose( size, h_a, h_c );

/* stop CPU timer */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  float elapsedTime;
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print CPU timing information */

  fprintf(stdout, "Total time CPU is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GB/s\n", 
    8.0 * 2.0 * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );

/* setup threadblock size and grid sizes */

  dim3 threads( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1 );
  dim3 blocks( ( size / THREADS_PER_BLOCK_X ) + 1, 
               ( size / THREADS_PER_BLOCK_Y ) + 1, 1 );

/* start timers */
  checkCUDA( hipEventRecord( start, 0 ) );

/* call smem GPU transpose kernel */

  smem_cuda_transpose<<< blocks, threads >>>( size, d_a, d_c );
  checkKERNEL()

/* stop the timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print GPU timing information */

  fprintf(stdout, "Total time GPU is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GB/s\n", 
    8.0 * 2.0 * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );

/* copy data from device to host */

  checkCUDA( hipMemset( d_a, 0, numbytes ) );
  checkCUDA( hipMemcpy( h_a, d_c, numbytes, hipMemcpyDeviceToHost ) );

/* compare GPU to CPU for correctness */

  int success = 1;
  
  for( int j = 0; j < size; j++ )
  {
    for( int i = 0; i < size; i++ )
    {
      if( h_c[INDX(i,j,size)] != h_a[INDX(i,j,size)] ) 
      {
        printf("Error in element %d,%d\n", i,j );
        printf("Host %f, device %f\n",h_c[INDX(i,j,size)],
                                      h_a[INDX(i,j,size)]);
        success = 0;
        break;
      }
    } /* end for i */
  } /* end for j */

  if( success == 1 ) printf("PASS\n");
  else               printf("FAIL\n");

/* free the memory */

  free( h_a );
  free( h_c );
  checkCUDA( hipFree( d_a ) );
  checkCUDA( hipFree( d_c ) );

  checkCUDA( hipDeviceReset() );

  return 0;
}
