#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "hipblas.h"
#include "../debug.h"

typedef float floatType_t;

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

/* define blocksize X and blocksize Y and blocksize K */

#define THREADS_PER_BLOCK_X 64 // Thread block size, x dimension
#define THREADS_PER_BLOCK_Y 64 // Thread block size, y dimension
#define BLOCK_K 64 // square block of K size

__global__ void GPU_shmem2(const int m, floatType_t const * const a, 
      floatType_t const * const b, floatType_t *c )
{

/* setup some constanst for later use */

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int iby = blockIdx.y * THREADS_PER_BLOCK_Y;
  const int ibx = blockIdx.x * THREADS_PER_BLOCK_X;

/* shared memory arrays for A and B */

  __shared__ floatType_t as[ THREADS_PER_BLOCK_X ][ BLOCK_K + 1 ];
  __shared__ floatType_t bs[ BLOCK_K ][ THREADS_PER_BLOCK_Y + 1 ];
	
/* space for C to be held in registers */

  floatType_t c_tmp = 0.0 ;

/* calculate my initial offset into A and B */

  int aoff = INDX( ibx + tx, ty, m );
  int boff = INDX( tx, iby + ty, m );

/* main loop over blocks of K */

  for( int Kblock = 0; Kblock < m; Kblock+=BLOCK_K )
  {

/* read block of A into shared memory */
    as[ tx ][ ty ] = a[ aoff ];

/* read block of B into shared memory */
    bs[ tx ][ ty ] = b[ boff ];
		
    __syncthreads();

/* increment A and B offsets  for next round of data reads */
    boff += BLOCK_K;
    aoff += m * BLOCK_K;

/* triply nested loop to perform the matmult on the blocks */

#pragma unroll
    for( int k = 0 ; k < BLOCK_K ; k++ )
    {
      c_tmp += as[ tx ][ k ] * bs[ k ][ ty ];
    }

    __syncthreads();

  } /* end for Kblock */

/* set C to its proper index int the C matrix */
  int coff = INDX( ibx + tx, iby + ty, m );

/* write results to the C matrix */
  c[ coff ] = c_tmp;
 
} /* end GPU_shmem2 */


int main( int argc, char *argv[] )
{

/* get GPU device number and name */
const int SIZE[1] =  {pow(2,10)};//, pow(2,16)};
for(size_t s = 0; s < 1; s++){
  const int size = SIZE[s];
  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );


  fprintf(stdout, "Matrix size is %d\n",size);

  floatType_t *h_a, *h_b, *h_c, *h_c1;
  floatType_t *d_a, *d_b, *d_c;
 
  size_t numbytes = (size_t ) size * (size_t ) size * sizeof( floatType_t );

  h_a = (floatType_t *) malloc( numbytes );
  if( h_a == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_b = (floatType_t *) malloc( numbytes );
  if( h_b == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_c = (floatType_t *) malloc( numbytes );
  if( h_c == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_c1 = (floatType_t *) malloc( numbytes );
  if( h_c1 == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

/* zero out the host memory for C matrices */

  memset( h_c, 0, numbytes );
  memset( h_c1, 0, numbytes );

  fprintf( stdout, "Total memory required is %lf MB\n", 
     3.0 * (double) numbytes / 1000000.0 );

/* initialize the A and B matrices */

  for( int i = 0; i < size * size; i++ )
  {
    h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
  }

/* allocate a, b, c in gpu memory */

  checkCUDA( hipMalloc( (void **)&d_a, numbytes ) );
  checkCUDA( hipMalloc( (void **)&d_b, numbytes ) );
  checkCUDA( hipMalloc( (void **)&d_c, numbytes ));
	
/* copy a and b to device */

  checkCUDA( hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice ) );
  checkCUDA( hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice ) );

  hipblasHandle_t handle;
  checkCUBLAS( hipblasCreate( &handle ) );

  floatType_t alpha = 1.0;
  floatType_t beta  = 0.0;

/* start timers */

  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );
  checkCUDA( hipEventRecord( start, 0 ) );

/* call CUBLAS dgemm */

  if( sizeof( floatType_t ) == 4 )
  {
  checkCUBLAS(
  hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               size, size, size,
               (float *)&alpha, 
               (float *)d_a, size,
               (float *)d_b, size,
               (float *)&beta,
               (float *)d_c, size )
             );
  } /* end if */
  else
  {
  checkCUBLAS(
  hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               size, size, size,
               (double *)&alpha, 
               (double *)d_a, size,
               (double *)d_b, size,
               (double *)&beta,
               (double *)d_c, size )
             );
  } /* end else */

/* stop timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  float elapsedTime;
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print GPU CUBLAS timing information */

  fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GFlop/s\n", 
    2.0 * (double) size * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
/* copy C from device to host for error checking */

  checkCUDA( hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost ) );

/* reset C on device to zero */

  checkCUDA( hipMemset( d_c, 0, numbytes ) );

/* setup grid and block sizes */

  dim3 threads( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1 );
  dim3 blocks( size / THREADS_PER_BLOCK_X, size / THREADS_PER_BLOCK_Y, 1 );

/* start timers */

  checkCUDA( hipEventRecord( start, 0 ) );

/* call GPU_naive */

  GPU_shmem2<<< blocks, threads >>> ( size, d_a, d_b, d_c );
  checkKERNEL()

/* stop timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print data for GPU naive */

  fprintf(stdout, "Total time GPU SHMEM is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GFlop/s\n", 
    2.0 * (double) size * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
/* copy C back to host */
	
  checkCUDA( hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost ) );

  checkCUBLAS( hipblasDestroy( handle ) );
  checkCUDA( hipEventDestroy( start ) );
  checkCUDA( hipEventDestroy( stop ) );

/* check CUBLAS versus GPU NAIVE numerical results */

  double temp = 0.0;

  for( int i = 0; i < size * size; i++ )
  {
     temp = max( temp, abs( (double)h_c[i] - (double)h_c1[i] )/
                      abs((double)h_c[i]) );
  } /* end for */
  printf("Maximum error is %e percent \n",temp*100.0);
  if( temp > 0.001 ) printf("FAIL\n");
  else printf("PASS\n");

/* cleanup */

  checkCUDA( hipFree( d_a ) );
  checkCUDA( hipFree( d_b ) );
  checkCUDA( hipFree( d_c ) );

  free( h_a );
  free( h_b );
  free( h_c );
  free( h_c1 );

  checkCUDA( hipDeviceReset() );
}
  return 0;
}
