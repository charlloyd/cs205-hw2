#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "../debug.h"

#define N ( 1 << 27 )
#define THREADS_PER_BLOCK 128

#define FLOATTYPE_T float

__global__ void sumReduction(int n, FLOATTYPE_T *in, FLOATTYPE_T *out)
{

  __shared__ FLOATTYPE_T sArray[THREADS_PER_BLOCK];

/* calculate global index in the array */
  int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
/* zero out the smem array */
  sArray[threadIdx.x] = 0.0;

/* grid stride handling case where array is larger than number of threads
 * launched
 * Loop over the grid stride so that each thread adds up its relevant 
 * elements of the array and saves them to SMEM
 */

  for( int i = globalIndex; i < n; i += blockDim.x * gridDim.x )
  {
    sArray[threadIdx.x] += in[i];
  } /* end for */
  
  __syncthreads();

/* do the final reduction in SMEM */
  for( int i = blockDim.x/2; i > 0; i = i / 2 )
  {
    if( threadIdx.x < i )
    {
      sArray[threadIdx.x] += sArray[threadIdx.x + i];
    } /* end if */
    __syncthreads();
  } /* end for */

/* thread0 of each threadblock writes the result to global memory */
  if( threadIdx.x == 0 ) out[blockIdx.x] = sArray[0]; 

  return;

}

int main()
{
  FLOATTYPE_T *h_in, h_sum, cpu_sum;
  FLOATTYPE_T *d_in, *d_sum, *d_tempArray;
  int size = N;
  int memBytes = size * sizeof( FLOATTYPE_T );
  int tempArraySize = 32768;

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

/* allocate space for device copies of in, out */

  checkCUDA( hipMalloc( &d_in, memBytes ) );
  checkCUDA( hipMalloc( &d_sum, sizeof(FLOATTYPE_T) ) );
  checkCUDA( hipMalloc( &d_tempArray, tempArraySize * sizeof(FLOATTYPE_T) ) );

/* allocate space for host copies of in, out and setup input values */

  h_in = (FLOATTYPE_T *)malloc( memBytes );

  for( int i = 0; i < size; i++ )
  {
    h_in[i] = FLOATTYPE_T( rand() ) / ( FLOATTYPE_T (RAND_MAX) + 1.0 );
    if( i % 2 == 0 ) h_in[i] = -h_in[i];
  }

  h_sum      = 0.0;
  cpu_sum   = 0.0;

/* copy inputs to device */

  checkCUDA( hipMemcpy( d_in, h_in, memBytes, hipMemcpyHostToDevice ) );
  checkCUDA( hipMemset( d_sum, 0, sizeof(FLOATTYPE_T) ) );
  checkCUDA( hipMemset( d_tempArray, 0, 
    tempArraySize * sizeof(FLOATTYPE_T) ) );

/* calculate block and grid sizes */

  dim3 threads1( THREADS_PER_BLOCK, 1, 1 );
  
  int blk = min( (size / threads1.x), tempArraySize );
  dim3 blocks( blk, 1, 1);

  dim3 threads2( min(blocks.x,threads1.x), 1, 1 );

/* start the timers */

  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );
  checkCUDA( hipEventRecord( start, 0 ) );

/* launch the kernel on the GPU */

  sumReduction<<< blocks, threads1 >>>( size, d_in,  d_tempArray );
  checkKERNEL()
  sumReduction<<<      1, threads2 >>>( blocks.x, d_tempArray, d_sum );
  checkKERNEL()

/* stop the timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  float elapsedTime;
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

  printf("Total elements is %d, %f GB\n", size, sizeof(FLOATTYPE_T)*
    (double)size * 1.e-9 );
  printf("GPU total time is %f ms, bandwidth %f GB/s\n", elapsedTime,
    sizeof(FLOATTYPE_T) * (double) size /
    ( (double) elapsedTime / 1000.0 ) * 1.e-9);

/* copy result back to host */

  checkCUDA( hipMemcpy( &h_sum, d_sum, sizeof(FLOATTYPE_T), 
    hipMemcpyDeviceToHost ) );

  checkCUDA( hipEventRecord( start, 0 ) );

  for( int i = 0; i < size; i++ )
  {
    cpu_sum += h_in[i];
  } /* end for */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );
  printf("CPU total time is %f ms, bandwidth %f GB/s\n", elapsedTime,
    sizeof(FLOATTYPE_T) * (double) size /
    ( (double) elapsedTime / 1000.0 ) * 1.e-9);


  FLOATTYPE_T diff = abs( cpu_sum - h_sum );

  if( diff / abs(h_sum) < 0.001 ) printf("PASS\n");
  else
  {                       
    printf("FAIL\n");
    printf("Error is %f\n", diff / h_sum );
    printf("GPU result is %f, CPU result is %f\n",h_sum, cpu_sum );
  } /* end else */

/* clean up */

  free(h_in);
  checkCUDA( hipFree( d_in ) );
  checkCUDA( hipFree( d_sum ) );
  checkCUDA( hipFree( d_tempArray ) );

  checkCUDA( hipDeviceReset() );
	
  return 0;
} /* end main */
