#include "hip/hip_runtime.h"
/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include "../debug.h"

#define N ( 1024 * 1024 )
#define RADIUS 5
#define THREADS_PER_BLOCK 64

/* stencil kernel */

__global__ void stencil_1d(int n, double *in, double *out)
{
/* allocate shared memory */
  __shared__ double temp[THREADS_PER_BLOCK + 2*(RADIUS)];

/* calculate global index in the array */
  int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int localIndex = threadIdx.x + RADIUS;

/* return if my global index is larger than the array size */
  if( globalIndex >= n ) return;

/* read input elements into shared memory */
  temp[localIndex] = in[globalIndex];

/* code to handle the halos.  need to make sure we don't walk off the end
   of the array */	
  if( threadIdx.x < RADIUS && globalIndex >= RADIUS )
  {
    temp[localIndex - RADIUS] = in[globalIndex - RADIUS];
  } /* end if */

  if( threadIdx.x < RADIUS && globalIndex < (n - RADIUS) )
  {
    temp[localIndex + THREADS_PER_BLOCK] = in[globalIndex + THREADS_PER_BLOCK];
  } /* end if */

  __syncthreads();
	
/* code to handle the boundary conditions */
  if( globalIndex < RADIUS || globalIndex >= (n - RADIUS) ) 
  {
    out[globalIndex] = (double) globalIndex * ( (double)RADIUS*2 + 1) ;
    return;
  } /* end if */

  double result = 0.0;

  for( int i = -(RADIUS); i <= (RADIUS); i++ ) 
  {
    result += temp[localIndex + i];
  } /* end for */

  out[globalIndex] = result;
  return;

}

int main()
{
  double *in, *out;
  double *d_in, *d_out;
  int size = N * sizeof( double );

/* get GPU device number and name */

  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

/* allocate space for device copies of in, out */

  checkCUDA( hipMalloc( (void **) &d_in, size ) );
  checkCUDA( hipMalloc( (void **) &d_out, size ) );

/* allocate space for host copies of in, out and setup input values */

  in = (double *)malloc( size );
  out = (double *)malloc( size );

  for( int i = 0; i < N; i++ )
  {
    in[i] = (double) i;
    out[i] = -99.0;
  }

/* copy inputs to device */

  checkCUDA( hipMemcpy( d_in, in, size, hipMemcpyHostToDevice ) );
  checkCUDA( hipMemset( d_out, 0, size ) );

/* calculate block and grid sizes */

  dim3 threads( THREADS_PER_BLOCK, 1, 1);
  dim3 blocks( (N / threads.x) + 1, 1, 1);

/* start the timers */

  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );
  checkCUDA( hipEventRecord( start, 0 ) );

/* launch the kernel on the GPU */

  stencil_1d<<< blocks, threads >>>( N, d_in, d_out );
  checkKERNEL()

/* stop the timers */

  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  float elapsedTime;
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );

  printf("Total time for %d elements was %f ms\n", N, elapsedTime );

/* copy result back to host */

  checkCUDA( hipMemcpy( out, d_out, size, hipMemcpyDeviceToHost ) );

  int success = 1;

  for( int i = 0; i < N; i++ )
  {
    if( in[i]*( (double)RADIUS*2+1 ) != out[i] )
    {
      printf("error in element %d in = %f out %f\n",i,in[i],out[i] );
      success = 0;
      break;
    } /* end if */
  } /* end for */

  if( success == 1 ) printf("PASS\n");
  else               printf("FAIL\n");

/* clean up */

  free(in);
  free(out);
  checkCUDA( hipFree( d_in ) );
  checkCUDA( hipFree( d_out ) );

  checkCUDA( hipDeviceSynchronize() );
	
  return 0;
} /* end main */
