/*
 *  Copyright 2017 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "headers.h"
#include "hipblas.h"
#include "kernels.h"

#define INDX(row,col,ld) (((col) * (ld)) + (row))

void svmTrain( floatType_t const *d_X, 
               floatType_t const *d_y, 
               floatType_t const C,
               int const numFeatures, int const numTrainingExamples,
               floatType_t const tol, int const maxPasses, 
               floatType_t *d_W )
{

/* declare pointers for arrays */
  floatType_t *d_K, *d_alphas, *d_f;

/* declare variables */
  floatType_t bHigh, bLow;
  int ILow, IHigh;

/* device variables */

  floatType_t *d_bLow, *d_bHigh;
  int *d_ILow, *d_IHigh;

/* cuBLAS data */
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  floatType_t const alpha=1.0; 
  floatType_t const beta =0.0;

/* malloc alphas */

  checkCUDA( hipMalloc( (void**) &d_alphas, 
               sizeof(floatType_t) * numTrainingExamples ) );

/* zero alphas */

  checkCUDA( hipMemset( d_alphas, 0, 
               sizeof(floatType_t)*numTrainingExamples ) );

/* malloc f */

  checkCUDA( hipMalloc( (void**) &d_f,
               sizeof(floatType_t) * numTrainingExamples ) );

  int threads_per_block = 256;
  k_initF<<<TRAINING_SET_SIZE/threads_per_block+1,threads_per_block>>>
                      ( d_f, d_y, numTrainingExamples );
  checkKERNEL()
/* malloc K, the kernel matrix */

  checkCUDA( hipMalloc( (void**) &d_K,
           sizeof(floatType_t) * numTrainingExamples * numTrainingExamples ) );
  checkCUDA( hipMemset( d_K, 0, 
           sizeof(floatType_t)*numTrainingExamples*numTrainingExamples ));

/* compute the Kernel on every pair of examples.
   K = X * X'
   Wouldn't do this in real life especially if X was really large.  
   For large K we'd just calculate the rows needed on the fly in the
   large loop
*/

  stat = hipblasCreate( &handle );
  if( stat != HIPBLAS_STATUS_SUCCESS ) printf("error creating cublas handle\n");

  if( sizeof( floatType_t ) == sizeof( float ) )
  {
    hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
               numTrainingExamples, numTrainingExamples, numFeatures,
               (float *)&alpha, (float *)d_X, numTrainingExamples,
               (float *)d_X, numTrainingExamples, (float *)&beta,
               (float *)d_K, numTrainingExamples );
   
  }
  else
  {
    hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
               numTrainingExamples, numTrainingExamples, numFeatures,
               (double *)&alpha, (double *)d_X, numTrainingExamples,
               (double *)d_X, numTrainingExamples, (double *)&beta,
               (double *)d_K, numTrainingExamples );
  }

  checkCUDA( hipMalloc( (void**)&d_bLow, sizeof(floatType_t) ) );
  checkCUDA( hipMalloc( (void**)&d_bHigh, sizeof(floatType_t) ) );
  checkCUDA( hipMalloc( (void**)&d_ILow, sizeof(int) ) );
  checkCUDA( hipMalloc( (void**)&d_IHigh, sizeof(int) ) );

  while( true )
  {

/* calculate the bLow and bHigh.  Must be called with only one 
   threadblock because it does a reduction.  Now what we'd do in practice
   but since the data size is small we can get away with it.
*/

    k_calculateBI<<<1,128>>>( d_f, d_alphas, d_y, numTrainingExamples,
                            d_bLow, d_bHigh, d_ILow, d_IHigh, C );
    checkKERNEL()

    checkCUDA( hipMemcpy( &bLow, d_bLow, sizeof(floatType_t),
                           hipMemcpyDeviceToHost ) );
    checkCUDA( hipMemcpy( &bHigh, d_bHigh, sizeof(floatType_t),
                           hipMemcpyDeviceToHost ) );
    checkCUDA( hipMemcpy( &ILow, d_ILow, sizeof(int),
                           hipMemcpyDeviceToHost ) );
    checkCUDA( hipMemcpy( &IHigh, d_IHigh, sizeof(int),
                           hipMemcpyDeviceToHost ) );

/* exit loop once we are below tolerance level */     
    if( bLow <= ( bHigh + ((floatType_t) 2.0 * tol) ) ) 
      break; 

/* update f array */

    k_updateF<<<TRAINING_SET_SIZE/threads_per_block + 1,threads_per_block>>>
                  ( d_f, d_alphas,
                           IHigh,
                           ILow,
                           d_K, numTrainingExamples, d_y, C, 
                           bLow, bHigh );
   checkKERNEL()

  } /* end while */

  k_scaleAlpha<<<TRAINING_SET_SIZE/threads_per_block + 1,threads_per_block>>>
                   ( d_alphas, d_y, numTrainingExamples );
  checkKERNEL()

/* calculate W from alphas */

  if( sizeof( floatType_t ) == sizeof( float ) )
  {
    hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               1, numFeatures, numTrainingExamples,
               (float *)&alpha, (float *)d_alphas, 1,
               (float *)d_X, numTrainingExamples, (float *)&beta,
               (float *)d_W, 1 );
  }
  else
  {
    hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               1, numFeatures, numTrainingExamples,
               (double *)&alpha, (double *)d_alphas, 1,
               (double *)d_X, numTrainingExamples, (double *)&beta,
               (double *)d_W, 1 );
  }
  
  checkCUDA( hipFree( d_alphas ) );
  checkCUDA( hipFree( d_f ) );
  checkCUDA( hipFree( d_K ) );
  checkCUDA( hipFree( d_ILow ) );
  checkCUDA( hipFree( d_IHigh ) );
  checkCUDA( hipFree( d_bLow ) );
  checkCUDA( hipFree( d_bHigh ) );

} /* end svmTrain */

void svmPredict( floatType_t const *X, 
                 floatType_t const *W, 
                 int const numExamples, int const numFeatures,
                 int *pred )
{
  floatType_t *p;

  p = (floatType_t *) malloc( sizeof(floatType_t) * numExamples );
  if( p == NULL ) fprintf(stderr,"error in malloc p in svmTrain\n");

  if( sizeof( floatType_t ) == 4 )
  {
    cblas_sgemv( CblasColMajor, CblasNoTrans,
               numExamples, numFeatures,
               1.0, (float *)X, numExamples,
               (float *)W, 1, 0.0,
               (float *)p, 1 );
  }
  else
  {
    cblas_dgemv( CblasColMajor, CblasNoTrans,
               numExamples, numFeatures,
               1.0, (double *)X, numExamples,
               (double *)W, 1, 0.0,
               (double *)p, 1 );
  }

  for( int i = 0; i < numExamples; i++ )
    pred[i] = ( p[i] >= 0.0 ) ? 1 : 0;
 
  free(p);
  return;
} /* end svmTrain */

void readMatrixFromFile( char *fileName, 
                         int *matrix, 
                         int const rows, 
                         int const cols )
{
  FILE *ifp;

  ifp = fopen( fileName, "r" );

  if( ifp == NULL ) 
  {
    fprintf(stderr, "Error opening file %s\n", fileName);
    exit(911);
  } /* end if */

  for( int row = 0; row < rows; row++ )
  {
    for( int col = 0; col < cols; col++ )
    {
      if( !fscanf( ifp, "%d", 
          &matrix[ INDX( row, col, rows ) ] ) )
      {
        fprintf(stderr,"error reading training matrix file \n");
        exit(911);
      } /* end if */
    } /* end for col */
  } /* end for row */

  fclose(ifp);
  return;
} /* end readMatrixFromFile */
