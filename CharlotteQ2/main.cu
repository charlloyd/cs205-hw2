
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel:
__global__ void square(float* numbers)
{
	// get the array coordinate:
	unsigned int x  = blockIdx.x * blockDim.x + threadIdx.x;
	
	// square the number:
	numbers[x] = numbers[x] * numbers[x];
}


// CPU Code:
int main (int argc, char const* argv[])
{
	const unsigned int N = 100;	// N numbers in array

	float data[N];		// array that contains numbers to be squared
	float squared[N];	// array to be filled with squared numbers
	
	// number to be squared will be the index:
	for(unsigned i=0; i<N; i++)
		data[i] = static_cast<float>(i);
	
	// allocate memory on CUDA device:
	float* pDevData;		// pointer to the data on the CUDA Device
	hipMalloc((void**)&pDevData, sizeof(data));
	
	// copy data to CUDA device:
	hipMemcpy(pDevData, &data, sizeof(data), hipMemcpyHostToDevice);
		
	// execute kernel function on GPU:
	square<<<10, 10>>>(pDevData);
	
	// copy data back from CUDA Device to 'squared' array:
	hipMemcpy(&squared, pDevData, sizeof(squared), hipMemcpyDeviceToHost);
	
	// free memory on the CUDA Device:
	hipFree(pDevData);
	
	// output results:
	for(unsigned i=0; i<N; i++)
		std::cout<<data[i]<<"^2 = "<<squared[i]<<"\n";
	
	return 0;
}
